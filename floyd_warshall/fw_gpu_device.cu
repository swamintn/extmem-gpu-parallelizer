#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "fw_gpu_common.h"

#define min(a, b) ((a) > (b))? (b): (a)

using namespace std;

/**
 * Encoding and decoding Morton codes
 * (Taken from https://fgiesen.wordpress.com/2009/12/13/decoding-morton-codes/)
 */
__host__ __device__ uint64_t encode2D_to_morton_64bit(uint64_t x, uint64_t y)
{
    x &= 0x00000000ffffffff;
    x = (x ^ (x <<  16)) & 0x0000ffff0000ffff;
    x = (x ^ (x <<  8))  & 0x00ff00ff00ff00ff;
    x = (x ^ (x <<  4))  & 0x0f0f0f0f0f0f0f0f;
    x = (x ^ (x <<  2))  & 0x3333333333333333;
    x = (x ^ (x <<  1))  & 0x5555555555555555;

    y &= 0x00000000ffffffff;
    y = (y ^ (y <<  16)) & 0x0000ffff0000ffff;
    y = (y ^ (y <<  8))  & 0x00ff00ff00ff00ff;
    y = (y ^ (y <<  4))  & 0x0f0f0f0f0f0f0f0f;
    y = (y ^ (y <<  2))  & 0x3333333333333333;
    y = (y ^ (y <<  1))  & 0x5555555555555555;

    // This will return row-major order z ordering. If we switch x and y, it will be column-major
    return (x << 1) | y;
}

/**
 * GPU memory allocator
 *
 * Allocates "pinned" host memory if type is pinned, else normal host memory
 */
void *mallocCudaHostMemory(unsigned int bytes, int type)
{
    void *memory;
    if (type == PINNED_HOST_MEMORY)
        hipHostMalloc((void**) &memory, bytes);
    else
        memory = malloc(bytes);
    return memory;
}

void freeCudaHostMemory(void *memory, int type)
{
    if (type == PINNED_HOST_MEMORY)
       hipHostFree(memory);
    else
        free(memory);
}

/*
 * GPU base case
 */
__device__ void parallel_base_case(unsigned long *X, unsigned long *U, unsigned long *V, uint64_t i, uint64_t j, uint64_t k)
{
    uint64_t cur = encode2D_to_morton_64bit(i, j);
    uint64_t first = encode2D_to_morton_64bit(i, k);
    uint64_t second = encode2D_to_morton_64bit(k, j);
    X[cur] = min(X[cur], (U[first] + V[second]));
}

/*
 * GPU Simple Kernel
 */
__global__ void parallel_iterative_kernel(unsigned long *X, unsigned long *U, unsigned long *V,
                                     uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol,
                                     uint64_t n)
{
    __shared__ unsigned long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;
 
    unsigned long *S_X = S_3X;
    unsigned long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    unsigned long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    // The entire array can fit in shared memory. Copy it here
    uint64_t X_off = encode2D_to_morton_64bit(xrow, xcol);
    uint64_t U_off = encode2D_to_morton_64bit(urow, ucol);
    uint64_t V_off = encode2D_to_morton_64bit(vrow, vcol);
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = U[U_off + (n*i + j)];
    S_V[n*i + j] = V[V_off + (n*i + j)];
    
    for (uint64_t k = 0; k < n; k++) {
        parallel_base_case(S_X, S_U, S_V, i, j, k);
        __syncthreads();
    }

    // Copy back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

/**
 * A_fw kernels
 */
__global__ void device_GPU_A_fw_A(unsigned long *X, uint64_t xrow, uint64_t xcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_X[ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;    
    
    // Copy to shared memory
    uint64_t off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*K);
    S_X[n*i + j] = X[off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        parallel_base_case(S_X, S_X, S_X, i, j, k);
        __syncthreads();
    }
    
    // Copy back to global memory
    X[off + (n*i + j)] = S_X[n*i + j];
}

#define B_ROW 0
#define C_ROW 1
__global__ void device_GPU_A_fw_B_C(unsigned long *X, uint64_t xrow, uint64_t xcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_2X[2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t B_or_C = blockIdx.x;
    uint64_t J_or_I = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_2X;
    unsigned long *S_U_or_V = S_2X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    
    uint64_t X_off;
    if (B_or_C == 0)
        X_off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*J_or_I);
    else
        X_off = encode2D_to_morton_64bit(xrow + n*J_or_I, xcol + n*K);
    uint64_t U_or_V_off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*K);
    
    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U_or_V[n*i + j]= X[U_or_V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        if (B_or_C == 0 && J_or_I != K)
            parallel_base_case(S_X, S_U_or_V, S_X, i, j, k);
        else if (B_or_C == 1 && J_or_I != K)
            parallel_base_case(S_X, S_X, S_U_or_V, i, j, k);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

__global__ void device_GPU_A_fw_D(unsigned long *X, uint64_t xrow, uint64_t xcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.x;
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_3X;
    unsigned long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    unsigned long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*J);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = X[U_off + (n*i + j)];
    S_V[n*i + j] = X[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        if (I != K && J != K)
            parallel_base_case(S_X, S_U, S_V, i, j, k);
        __syncthreads();
    }
    
    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];   
}

/**
 * Host GPU code - Launcher for A_fw kernels
 */
void host_GPU_A_fw(unsigned long *X,
                   uint64_t xrow, uint64_t xcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_SHARED) {
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        parallel_iterative_kernel<<<1, threads_per_block>>>(X, X, X, xrow, xcol, xrow, xcol, xrow, xcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_SHARED;
        uint64_t m = n / r;
        // DEBUG: cout << "A: Splitting GPU global matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        dim3 blocks_per_grid_A(1);
        dim3 blocks_per_grid_B_C(2, r);
        dim3 blocks_per_grid_D(r, r);

        for (uint64_t k = 0; k < r; k++) {
            // Step 1: A_step - A(X_kk, U_kk, V_kk), X,U,V are the same
            device_GPU_A_fw_A<<<blocks_per_grid_A, threads_per_block>>>(X, xrow, xcol, k, m);
            
            // Step 2: B_C_step - B(X_kj, U_kk, V_kj), C(X_ik, U_ik, V_kk)
            device_GPU_A_fw_B_C<<<blocks_per_grid_B_C, threads_per_block>>>(X, xrow, xcol, k, m);

            // Step 3: D_step - D_step - D(X_ij, U_ik, V_kj)
            device_GPU_A_fw_D<<<blocks_per_grid_D, threads_per_block>>>(X, xrow, xcol, k, m);
        }
    }
}

/** 
 * B_fw kernels
 */
__global__ void device_GPU_B_fw_B(unsigned long *X, unsigned long *U,
                                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_2X[2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_2X;
    unsigned long *S_U = S_2X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(urow + n*K, ucol + n*K);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = U[U_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        parallel_base_case(S_X, S_U, S_X, i, j, k);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];    
}

__global__ void device_GPU_B_fw_D(unsigned long *X, unsigned long *U,
                                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.x;
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_3X;
    unsigned long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    unsigned long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(urow + n*I, ucol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(xrow + n*K, xcol + n*J);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = U[U_off + (n*i + j)];
    S_V[n*i + j] = X[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        if (I != K)
            parallel_base_case(S_X, S_U, S_V, i, j, k);
        __syncthreads();
    }
    
    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

/**
 * Host GPU code - Launcher for B_fw kernels
 */
void host_GPU_B_fw(unsigned long *X, unsigned long *U,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_SHARED) {
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        parallel_iterative_kernel<<<1, threads_per_block>>>(X, U, X, xrow, xcol, urow, ucol, xrow, xcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_SHARED;
        uint64_t m = n / r;
        // DEBUG: cout << "B: Splitting GPU global matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        dim3 blocks_per_grid_B(1, r);
        dim3 blocks_per_grid_D(r, r);
        
        for (uint64_t k = 0; k < r; k++) {
            // Step 1: B_step - B(X_kj, U_kk, Vkj)
            device_GPU_B_fw_B<<<blocks_per_grid_B, threads_per_block>>>(X, U, xrow, xcol, urow, ucol, k, m);

            // Step 2: D_step - D(X_ij, U_ik, V_kj)
            device_GPU_B_fw_D<<<blocks_per_grid_D, threads_per_block>>>(X, U, xrow, xcol, urow, ucol, k, m);
        }
    }
}

/**
 * C_fw kernels
 */
__global__ void device_GPU_C_fw_C(unsigned long *X, unsigned long *V,
                                  uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_2X[2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_2X;
    unsigned long *S_V = S_2X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(vrow + n*K, vcol + n*K);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_V[n*i + j] = V[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        parallel_base_case(S_X, S_X, S_V, i, j, k);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

__global__ void device_GPU_C_fw_D(unsigned long *X, unsigned long *V,
                                  uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.x;
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_3X;
    unsigned long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    unsigned long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(vrow + n*K, vcol + n*J);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = X[U_off + (n*i + j)];
    S_V[n*i + j] = V[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        if (J != K)
            parallel_base_case(S_X, S_U, S_V, i, j, k);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}


/**
 * Host GPU code - Launcher for C_fw kernels
 */
void host_GPU_C_fw(unsigned long *X, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_SHARED) {
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        parallel_iterative_kernel<<<1, threads_per_block>>>(X, X, V, xrow, xcol, xrow, xcol, vrow, vcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_SHARED;
        uint64_t m = n / r;
        // DEBUG: cout << "C: Splitting GPU global matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        dim3 blocks_per_grid_C(1, r);
        dim3 blocks_per_grid_D(r, r);
        
        for (uint64_t k = 0; k < r; k++) {
            // Step 1: B_step - C(X_ik, U_kk, V_kk)
            device_GPU_C_fw_C<<<blocks_per_grid_C, threads_per_block>>>(X, V, xrow, xcol, vrow, vcol, k, m);

            // Step 2: D_step - D(X_ij, U_ik, V_kj)
            device_GPU_C_fw_D<<<blocks_per_grid_D, threads_per_block>>>(X, V, xrow, xcol, vrow, vcol, k, m);
        }
    }
}

/**
 * D_fw kernels
 */
__global__ void device_GPU_D_fw_D(unsigned long *X, unsigned long *U, unsigned long *V,
                                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t K, uint64_t n)
{
    __shared__ unsigned long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.x;
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    unsigned long *S_X = S_3X;
    unsigned long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    unsigned long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(urow + n*I, ucol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(vrow + n*K, vcol + n*J);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = U[U_off + (n*i + j)];
    S_V[n*i + j] = V[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        parallel_base_case(S_X, S_U, S_V, i, j, k);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

/**
 * Host GPU code - Launcher for D_fw kernels
 */
void host_GPU_D_fw(unsigned long *X, unsigned long *U, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_SHARED) {
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        parallel_iterative_kernel<<<1, threads_per_block>>>(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_SHARED;
        uint64_t m = n / r;
        // DEBUG: cout << "D: Splitting GPU global matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        dim3 blocks_per_grid_D(r, r);
        
        for (uint64_t k = 0; k < r; k++) {
            // Step 1: D_step - D(X_ij, U_ik, V_kj)
            device_GPU_D_fw_D<<<blocks_per_grid_D, threads_per_block>>>(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, k, m);
        }
    }
}

/*
 * Host RAM code
 */
void host_RAM_A_fw(unsigned long *X,
                   uint64_t xrow, uint64_t xcol, uint64_t n)
{
    // Base case - If possible, read the entire array into GPU
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        unsigned long *G_X = NULL;
        unsigned int bytes = n * n * sizeof(unsigned long);
        // DEBUG: cout << "Reached RAM_A base case, size=" << n << endl;       
        hipMalloc(&G_X, bytes);
        hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow, xcol), bytes, hipMemcpyHostToDevice);
        host_GPU_A_fw(G_X, 0, 0, n);
        hipMemcpy(X + encode2D_to_morton_64bit(xrow, xcol), G_X, bytes, hipMemcpyDeviceToHost);
        hipFree(G_X);
    }
    // If not, split into r chunks
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        // DEBUG: cout << "A_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        unsigned long *G_W  = NULL;
        unsigned long *G_R1 = NULL;
        unsigned long *G_R2 = NULL;
        unsigned int bytes = m * m * sizeof(unsigned long);

        hipMalloc(&G_W, bytes);
        hipMalloc(&G_R1, bytes);
        hipMalloc(&G_R2, bytes);
        
        for (uint64_t k = 0; k < r; k++) {
            // Step 1: A_step - A(X_kk, U_kk, V_kk), X,U,V are the same
            hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*k), bytes, hipMemcpyHostToDevice);
            host_GPU_A_fw(G_W, 0, 0, m);
            // Keep this in device memory itself since B and C can use it. Take care to write it to RAM later
            
            // Step 2: B_C_step - B(X_kj, U_kk, V_kj), C(X_ik, U_ik, V_kk)
            // Note that B's U_kk and C's V_kk are the same
            // For B, X and V are the same, for C, X and U are the same.
            // We already have B's U_kk/C's V_kk in device memory, put it in R1
            unsigned long *G_T = G_R1; G_R1 = G_W; G_W = G_T;
            for (uint64_t j = 0; j < r; j++) {
                if (j != k) {
                    hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), bytes, hipMemcpyHostToDevice);
                    host_GPU_B_fw(G_W, G_R1, 0, 0, 0, 0, m);
                    hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);                
                }
            }
            for (uint64_t i = 0; i < r; i++) {
                if (i != k) {
                    hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), bytes, hipMemcpyHostToDevice);
                    host_GPU_C_fw(G_W, G_R1, 0, 0, 0, 0, m);
                    hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), G_W, bytes, hipMemcpyDeviceToHost);
                }
            }
            // Write the deferred A-step X_kk to RAM
            hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*k), G_R1, bytes, hipMemcpyDeviceToHost);

            // Step 3: D_step - D(X_ij, U_ik, V_kj)           
            for (uint64_t i = 0; i < r; i++) {
                if (i != k) {
                    // U_ik is same for all j
                    hipMemcpy(G_R1, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), bytes, hipMemcpyHostToDevice);
                    for (uint64_t j = 0; j < r; j++) {
                        if (j != k) {
                            hipMemcpy(G_R2, X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), bytes, hipMemcpyHostToDevice);
                            hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), bytes, hipMemcpyHostToDevice);
                            host_GPU_D_fw(G_W, G_R1, G_R2, 0, 0, 0, 0, 0, 0, m);
                            hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);
                        }
                    }
                }
            }
        }       
        hipFree(G_W);
        hipFree(G_R1);
        hipFree(G_R2);
    }
}

void host_RAM_B_fw(unsigned long *X, unsigned long *U,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        unsigned long *G_X = NULL;
        unsigned long *G_U = NULL;
        unsigned int bytes = n * n * sizeof(unsigned long);
        // DEBUG: cout << "Reached RAM_B base case, size=" << n << endl;       
        hipMalloc(&G_X, bytes);
        hipMalloc(&G_U, bytes);
        hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow, xcol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_U, U + encode2D_to_morton_64bit(urow, ucol), bytes, hipMemcpyHostToDevice);
        host_GPU_B_fw(G_X, G_U, 0, 0, 0, 0, n);
        hipMemcpy(X + encode2D_to_morton_64bit(xrow, xcol), G_X, bytes, hipMemcpyDeviceToHost);
        hipFree(G_X);
        hipFree(G_U);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        // DEBUG: cout << "B_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        unsigned long *G_W  = NULL;
        unsigned long *G_R1 = NULL;
        unsigned long *G_R2 = NULL;
        unsigned int bytes = m * m * sizeof(unsigned long);

        hipMalloc(&G_W, bytes);
        hipMalloc(&G_R1, bytes);
        hipMalloc(&G_R2, bytes);

        for (uint64_t k = 0; k < r; k++) {
            hipMemcpy(G_R1, U + encode2D_to_morton_64bit(urow + m*k, ucol + m*k), bytes, hipMemcpyHostToDevice);
            for (uint64_t j = 0; j < r; j++) {
                hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), bytes, hipMemcpyHostToDevice);
                host_GPU_B_fw(G_W, G_R1, 0, 0, 0, 0, m);
                hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);
            }
            for (uint64_t i = 0; i < r; i++) {
                hipMemcpy(G_R1, U + encode2D_to_morton_64bit(urow + m*i, ucol + m*k), bytes, hipMemcpyHostToDevice);
                for (uint64_t j = 0; j < r; j++) {
                    if (i != k) {
                        hipMemcpy(G_R2, X + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), bytes, hipMemcpyHostToDevice);
                        hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), bytes, hipMemcpyHostToDevice);
                        host_GPU_D_fw(G_W, G_R1, G_R2, 0, 0, 0, 0, 0, 0, m);
                        hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);
                    }
                }
            }
        }
        hipFree(G_W);
        hipFree(G_R1);
        hipFree(G_R2);
    }
}

void host_RAM_C_fw(unsigned long *X, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        unsigned long *G_X = NULL;
        unsigned long *G_V = NULL;
        unsigned int bytes = n * n * sizeof(unsigned long);
        // DEBUG: cout << "Reached RAM_C base case, size=" << n << endl;
        hipMalloc(&G_X, bytes);
        hipMalloc(&G_V, bytes);
        hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow, xcol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_V, V + encode2D_to_morton_64bit(vrow, vcol), bytes, hipMemcpyHostToDevice);
        host_GPU_C_fw(G_X, G_V, 0, 0, 0, 0, n);
        hipMemcpy(X + encode2D_to_morton_64bit(xrow, xcol), G_X, bytes, hipMemcpyDeviceToHost);
        hipFree(G_X);
        hipFree(G_V);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        // DEBUG: cout << "C_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        unsigned long *G_W  = NULL;
        unsigned long *G_R1 = NULL;
        unsigned long *G_R2 = NULL;
        unsigned int bytes = m * m * sizeof(unsigned long);

        hipMalloc(&G_W, bytes);
        hipMalloc(&G_R1, bytes);
        hipMalloc(&G_R2, bytes);

        for (uint64_t k = 0; k < r; k++) {
            hipMemcpy(G_R1, V + encode2D_to_morton_64bit(vrow + m*k, vcol + m*k), bytes, hipMemcpyHostToDevice);
            for (uint64_t i = 0; i < r; i++) {
                hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), bytes, hipMemcpyHostToDevice);
                host_GPU_C_fw(G_W, G_R1, 0, 0, 0, 0, m);
                hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), G_W, bytes, hipMemcpyDeviceToHost);
            }
            for (uint64_t i = 0; i < r; i++) {
                hipMemcpy(G_R1, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*k), bytes, hipMemcpyHostToDevice);
                for (uint64_t j = 0; j < r; j++) {
                    if (j != k) {
                        hipMemcpy(G_R2, V + encode2D_to_morton_64bit(xrow + m*k, xcol + m*j), bytes, hipMemcpyHostToDevice);
                        hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), bytes, hipMemcpyHostToDevice);
                        host_GPU_D_fw(G_W, G_R1, G_R2, 0, 0, 0, 0, 0, 0, m);
                        hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);
                    }
                }
            }
        }
        hipFree(G_W);
        hipFree(G_R1);
        hipFree(G_R2);
    }
}

void host_RAM_D_fw(unsigned long *X, unsigned long *U, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        unsigned long *G_X = NULL;
        unsigned long *G_U = NULL;
        unsigned long *G_V = NULL;
        unsigned int bytes = n * n * sizeof(unsigned long);
        // DEBUG: cout << "Reached RAM_D base case, size=" << n << endl;
        hipMalloc(&G_X, bytes);
        hipMalloc(&G_U, bytes);
        hipMalloc(&G_V, bytes);
        hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow, xcol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_U, U + encode2D_to_morton_64bit(urow, ucol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_V, V + encode2D_to_morton_64bit(vrow, vcol), bytes, hipMemcpyHostToDevice);
        host_GPU_D_fw(G_X, G_U, G_V, 0, 0, 0, 0, 0, 0, n);
        hipMemcpy(X + encode2D_to_morton_64bit(xrow, xcol), G_X, bytes, hipMemcpyDeviceToHost);
        hipFree(G_X);
        hipFree(G_U);
        hipFree(G_V);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        // DEBUG: cout << "D_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        unsigned long *G_W  = NULL;
        unsigned long *G_R1 = NULL;
        unsigned long *G_R2 = NULL;
        unsigned int bytes = m * m * sizeof(unsigned long);

        hipMalloc(&G_W, bytes);
        hipMalloc(&G_R1, bytes);
        hipMalloc(&G_R2, bytes);

        for (uint64_t k = 0; k < r; k++) {
            for (uint64_t i = 0; i < r; i++) {
                hipMemcpy(G_R1, U + encode2D_to_morton_64bit(urow + m*i, ucol + m*k), bytes, hipMemcpyHostToDevice);
                for (uint64_t j = 0; j < r; j++) {
                    hipMemcpy(G_R2, V + encode2D_to_morton_64bit(vrow + m*k, vcol + m*j), bytes, hipMemcpyHostToDevice);
                    hipMemcpy(G_W, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), bytes, hipMemcpyHostToDevice);
                    host_GPU_D_fw(G_W, G_R1, G_R2, 0, 0, 0, 0, 0, 0, m);
                    hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), G_W, bytes, hipMemcpyDeviceToHost);
                }
            }
        }
        hipFree(G_W);
        hipFree(G_R1);
        hipFree(G_R2);
    }
}

