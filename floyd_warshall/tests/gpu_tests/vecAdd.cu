#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;

/**
 * GPU Kernel
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

int main()
{
	// Error code checking
	hipError_t err = hipSuccess;

	int numElements = 50000;
	size_t size = numElements * sizeof(float);
	cout << "Vector addition of " << numElements << " elements\n...";

	// Host vectors
	float *h_A = (float *)malloc(size);
	float *h_B = (float *)malloc(size);
	float *h_C = (float *)malloc(size);
	if (h_A == NULL || h_B == NULL || h_C == NULL)
	{
		cout << "ERROR: Host vector memory allocation failed\n";
		exit(EXIT_FAILURE);
	}

	// Initialize host vectors
	for (int i = 0; i < numElements; ++i) {
		h_A[i] = (i + 1);
		h_B[i] = (i + 1) * 2;
	}

	// Device vectors
	float *d_A = NULL;
	float *d_B = NULL;
	float *d_C = NULL;
	hipMalloc(&d_A, size);	
	hipMalloc(&d_B, size);	
	hipMalloc(&d_C, size);
	
	// Copy memory from host to device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// Kernel properties
	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	cout << "Launching kernel with " << blocksPerGrid << " blocks and " << threadsPerBlock << " threads...\n";
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
	cout << "ERROR: Failed to launch vectorAdd kernel (error is '" << hipGetErrorString(err) << "')\n";
        exit(EXIT_FAILURE);
	}

	// Copy memory from device to host
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// Print vector
	for (int i = 0; i < numElements; ++i) {
		cout << "h_A[" << i << "] = " << h_A[i] << ", h_A[" << i << "] = " << h_B[i] << ", h_C[ " << i << "] = " << h_C[i] << endl;
	} 

}
