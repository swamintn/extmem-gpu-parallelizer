#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "fw_gpu_common.h"

using namespace std;

/**
 * GPU memory allocator
 *
 * Allocates "pinned" host memory if type is pinned, else normal host memory
 */
void *mallocCudaHostMemory(unsigned int bytes, int type)
{
    void *memory;
    if (type == PINNED_HOST_MEMORY)
        hipHostMalloc((void**) &memory, bytes);
    else
        memory = malloc(bytes);
    return memory;
}

void freeCudaHostMemory(void *memory, int type)
{
    if (type == PINNED_HOST_MEMORY)
       hipHostFree(memory);
    else
        free(memory);
}

/**
 * Host GPU code
 */
void host_GPU_A_fw(unsigned long *X,
                   uint64_t xrow, uint64_t xcol, uint64_t n){}
void host_GPU_B_fw(unsigned long *X, unsigned long *U,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t n){}
void host_GPU_C_fw(unsigned long *X, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t n){}
void host_GPU_D_fw(unsigned long *X, unsigned long *U, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n){}

/*
 * Host RAM code
 */
void host_RAM_A_fw(unsigned long *X,
                   uint64_t xrow, uint64_t xcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        host_GPU_A_fw(X, xrow, xcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        cout << "A_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        for (uint64_t k = 0; k < r; k++) {
            host_GPU_A_fw(X, xrow + m*k, xcol + m*k, m);
            for (uint64_t j = 0; j < r; j++) {
                if (j != k) {
                    host_GPU_B_fw(X, X, xrow + m*k, xcol + m*j, xrow + m*k, xcol + m*k, m);
                }
            }
            for (uint64_t i = 0; i < r; i++) {
                if (i != k) {
                    host_GPU_C_fw(X, X, xrow + m*i, xcol + m*k, xrow + m*k, xcol + m*k, m);
                }
            }
            for (uint64_t i = 0; i < r; i++) {
                for (uint64_t j = 0; j < r; j++) {
                    if (i != k && j != k) {
                        host_GPU_D_fw(X, X, X, xrow + m*i, xcol + m*j, xrow + m*i, xcol + m*k, xrow + m*k, xcol + m*j, m);
                    }
                }
            }
        }
    }
}

void host_RAM_B_fw(unsigned long *X, unsigned long *U,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        host_GPU_B_fw(X, U, xrow, xcol, urow, ucol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        cout << "B_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        for (uint64_t k = 0; k < r; k++) {
            for (uint64_t j = 0; j < r; j++) {
                host_GPU_B_fw(X, U, xrow + m*k, xcol + m*j, urow + m*k, ucol + m*k, m);
            }
            for (uint64_t i = 0; i < r; i++) {
                for (uint64_t j = 0; j < r; j++) {
                    if (i != k) {
                        host_GPU_D_fw(X, U, X, xrow + m*i, xcol + m*j, urow + m*i, ucol + m*k, xrow + m*k, xcol + m*j, m);
                    }
                }
            }
        }
    }
}

void host_RAM_C_fw(unsigned long *X, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        host_GPU_C_fw(X, V, xrow, xcol, vrow, vcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        cout << "C_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        for (uint64_t k = 0; k < r; k++) {
            for (uint64_t i = 0; i < r; i++) {
                host_GPU_C_fw(X, V, xrow + m*i, xcol + m*k, vrow + m*k, vcol + m*k, m);
            }
            for (uint64_t i = 0; i < r; i++) {
                for (uint64_t j = 0; j < r; j++) {
                    if (j != k) {
                        host_GPU_D_fw(X, X, V, xrow + m*i, xcol + m*j, xrow + m*i, xcol + m*k, vrow + m*k, vcol + m*j, m);
                    }
                }
            }
        }
    }
}

void host_RAM_D_fw(unsigned long *X, unsigned long *U, unsigned long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        host_GPU_D_fw(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        cout << "D_fw: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        for (uint64_t k = 0; k < r; k++) {
            for (uint64_t i = 0; i < r; i++) {
                for (uint64_t j = 0; j < r; j++) {
                    host_GPU_D_fw(X, U, V, xrow + m*i, xcol + m*j, urow + m*i, ucol + m*k, vrow + m*k, vcol + m*j, m);
                }
            }
        }
    }
}


