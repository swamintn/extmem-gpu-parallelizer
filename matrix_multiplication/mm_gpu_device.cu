#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "mm_gpu_common.h"

#define min(a, b) ((a) > (b))? (b): (a)

using namespace std;

/**
 * Encoding and decoding Morton codes
 * (Taken from https://fgiesen.wordpress.com/2009/12/13/decoding-morton-codes/)
 */
__host__ __device__ uint64_t encode2D_to_morton_64bit(uint64_t x, uint64_t y)
{
    x &= 0x00000000ffffffff;
    x = (x ^ (x <<  16)) & 0x0000ffff0000ffff;
    x = (x ^ (x <<  8))  & 0x00ff00ff00ff00ff;
    x = (x ^ (x <<  4))  & 0x0f0f0f0f0f0f0f0f;
    x = (x ^ (x <<  2))  & 0x3333333333333333;
    x = (x ^ (x <<  1))  & 0x5555555555555555;

    y &= 0x00000000ffffffff;
    y = (y ^ (y <<  16)) & 0x0000ffff0000ffff;
    y = (y ^ (y <<  8))  & 0x00ff00ff00ff00ff;
    y = (y ^ (y <<  4))  & 0x0f0f0f0f0f0f0f0f;
    y = (y ^ (y <<  2))  & 0x3333333333333333;
    y = (y ^ (y <<  1))  & 0x5555555555555555;

    // This will return row-major order z ordering. If we switch x and y, it will be column-major
    return (x << 1) | y;
}

/**
 * GPU memory allocator
 *
 * Allocates "pinned" host memory if type is pinned, else normal host memory
 */
void *mallocCudaHostMemory(unsigned int bytes, int type)
{
    void *memory;
    if (type == PINNED_HOST_MEMORY)
        hipHostMalloc((void**) &memory, bytes);
    else
        memory = malloc(bytes);
    return memory;
}

void freeCudaHostMemory(void *memory, int type)
{
    if (type == PINNED_HOST_MEMORY)
       hipHostFree(memory);
    else
        free(memory);
}

/**
 * D_mm kernels
 */
__global__ void device_GPU_A_mm(long *X, long *U, long *V,
                                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t K, uint64_t n)
{
    __shared__ long S_3X[3*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED];
    uint64_t I = blockIdx.x;
    uint64_t J = blockIdx.y;
    uint64_t i = threadIdx.x;
    uint64_t j = threadIdx.y;

    long *S_X = S_3X;
    long *S_U = S_3X + (ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);
    long *S_V = S_3X + (2*ALLOWED_SIZE_GPU_SHARED*ALLOWED_SIZE_GPU_SHARED);

    uint64_t X_off = encode2D_to_morton_64bit(xrow + n*I, xcol + n*J);
    uint64_t U_off = encode2D_to_morton_64bit(urow + n*I, ucol + n*K);
    uint64_t V_off = encode2D_to_morton_64bit(vrow + n*K, vcol + n*J);

    // Copy to shared memory
    S_X[n*i + j] = X[X_off + (n*i + j)];
    S_U[n*i + j] = U[U_off + (n*i + j)];
    S_V[n*i + j] = V[V_off + (n*i + j)];
    __syncthreads();

    for (uint64_t k = 0; k < n; k++) {
        uint64_t cur = encode2D_to_morton_64bit(i, j);
        uint64_t first = encode2D_to_morton_64bit(i, k);
        uint64_t second = encode2D_to_morton_64bit(k, j);
        S_X[cur] += (S_U[first] * S_V[second]);
        __syncthreads();
    }

    // Copy only X back to global memory
    X[X_off + (n*i + j)] = S_X[n*i + j];
}

/**
 * Host GPU code - Launcher for the kernel
 */
void host_GPU_A_mm(long *X, long *U, long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_SHARED) {
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        // This is like r-way with r = 1. So k loop through only one value - '0'
        device_GPU_A_mm<<<1, threads_per_block>>>(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, 0, n);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_SHARED;
        uint64_t m = n / r;
        // DEBUG: cout << "A: Splitting GPU global matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;
        
        dim3 threads_per_block(ALLOWED_SIZE_GPU_SHARED, ALLOWED_SIZE_GPU_SHARED);
        dim3 blocks_per_grid_A(r, r);
        
        for (uint64_t k = 0; k < r; k++) {
            // Step 1: A_step - D(X_ij, U_ik, V_kj)
            device_GPU_A_mm<<<blocks_per_grid_A, threads_per_block>>>(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, k, m);
        }
    }
}

/*
 * Host RAM code
 */
void host_RAM_A_mm(long *X, long *U, long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    if (n <= ALLOWED_SIZE_GPU_GLOBAL) {
        long *G_X = NULL;
        long *G_U = NULL;
        long *G_V = NULL;
        unsigned int bytes = n * n * sizeof(long);
        // DEBUG: cout << "Reached RAM_A base case, size=" << n << endl;
        hipMalloc(&G_X, bytes);
        hipMalloc(&G_U, bytes);
        hipMalloc(&G_V, bytes);
        hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow, xcol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_U, U + encode2D_to_morton_64bit(urow, ucol), bytes, hipMemcpyHostToDevice);
        hipMemcpy(G_V, V + encode2D_to_morton_64bit(vrow, vcol), bytes, hipMemcpyHostToDevice);
        host_GPU_A_mm(G_X, G_U, G_V, 0, 0, 0, 0, 0, 0, n);
        hipMemcpy(X + encode2D_to_morton_64bit(xrow, xcol), G_X, bytes, hipMemcpyDeviceToHost);
        hipFree(G_X);
        hipFree(G_U);
        hipFree(G_V);
    }
    else {
        uint64_t r = n / ALLOWED_SIZE_GPU_GLOBAL;
        uint64_t m = n / r;
        // DEBUG: cout << "A_mm: Splitting RAM matrix into r=" << r << " chunks, each submatrix size, m=" << m << endl;

        long *G_X  = NULL;
        long *G_U = NULL;
        long *G_V = NULL;
        unsigned int bytes = m * m * sizeof(long);

        hipMalloc(&G_X, bytes);
        hipMalloc(&G_U, bytes);
        hipMalloc(&G_V, bytes);

        for (uint64_t k = 0; k < r; k++) {
            for (uint64_t i = 0; i < r; i++) {
                hipMemcpy(G_U, U + encode2D_to_morton_64bit(urow + m*i, ucol + m*k), bytes, hipMemcpyHostToDevice);
                for (uint64_t j = 0; j < r; j++) {
                    hipMemcpy(G_V, V + encode2D_to_morton_64bit(vrow + m*k, vcol + m*j), bytes, hipMemcpyHostToDevice);
                    hipMemcpy(G_X, X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), bytes, hipMemcpyHostToDevice);
                    host_GPU_A_mm(G_X, G_U, G_V, 0, 0, 0, 0, 0, 0, m);
                    hipMemcpy(X + encode2D_to_morton_64bit(xrow + m*i, xcol + m*j), G_X, bytes, hipMemcpyDeviceToHost);
                }
            }
        }
        hipFree(G_X);
        hipFree(G_U);
        hipFree(G_V);
    }
}


/*
 * Serial Matrix Multiplication code (for verification if necessary)
 */
void serial_mm(long *X, long *U, long *V,
                   uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    cout << "Serial for n= " << n << endl;
    for (uint64_t k = 0; k < n; k++) {
        for (uint64_t i = 0; i < n; i++) {
            for (uint64_t j = 0; j < n; j++) {
                uint64_t U_ind = encode2D_to_morton_64bit(urow + i, ucol + k);
                uint64_t V_ind = encode2D_to_morton_64bit(vrow + k, vcol + j);
                uint64_t X_ind = encode2D_to_morton_64bit(xrow + i, xcol + j);
                X[X_ind] += (U[U_ind] * V[V_ind]);
            }
        }
    }       
}

