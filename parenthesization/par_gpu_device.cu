#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "par_gpu_common.h"

#define min(a, b) ((a) > (b))? (b): (a)

using namespace std;

/**
 * Encoding and decoding Morton codes
 * (Taken from https://fgiesen.wordpress.com/2009/12/13/decoding-morton-codes/)
 */
__host__ __device__ uint64_t encode2D_to_morton_64bit(uint64_t x, uint64_t y)
{
    x &= 0x00000000ffffffff;
    x = (x ^ (x <<  16)) & 0x0000ffff0000ffff;
    x = (x ^ (x <<  8))  & 0x00ff00ff00ff00ff;
    x = (x ^ (x <<  4))  & 0x0f0f0f0f0f0f0f0f;
    x = (x ^ (x <<  2))  & 0x3333333333333333;
    x = (x ^ (x <<  1))  & 0x5555555555555555;

    y &= 0x00000000ffffffff;
    y = (y ^ (y <<  16)) & 0x0000ffff0000ffff;
    y = (y ^ (y <<  8))  & 0x00ff00ff00ff00ff;
    y = (y ^ (y <<  4))  & 0x0f0f0f0f0f0f0f0f;
    y = (y ^ (y <<  2))  & 0x3333333333333333;
    y = (y ^ (y <<  1))  & 0x5555555555555555;

    // This will return row-major order z ordering. If we switch x and y, it will be column-major
    return (x << 1) | y;
}

/**
 * GPU memory allocator
 *
 * Allocates "pinned" host memory if type is pinned, else normal host memory
 */
void *mallocCudaHostMemory(unsigned int bytes, int type)
{
    void *memory;
    if (type == PINNED_HOST_MEMORY)
        hipHostMalloc((void**) &memory, bytes);
    else
        memory = malloc(bytes);
    return memory;
}

void freeCudaHostMemory(void *memory, int type)
{
    if (type == PINNED_HOST_MEMORY)
       hipHostFree(memory);
    else
        free(memory);
}

/*
 * Serial code
 */
void serial_A_par(unsigned long *X,
                  uint64_t xrow, uint64_t xcol, uint64_t n)
{
    for (int64_t t = 2; t <= n-1; t++) {
        for (int64_t i = 0; i <= n - t - 1; i++) {
            int64_t j = t + i;
            for (int64_t k = i + 1; k <= j; k++) {
                uint64_t cur = encode2D_to_morton_64bit(xrow + i, xcol + j);
                uint64_t first = encode2D_to_morton_64bit(xrow + i, xcol + k);
                uint64_t second = encode2D_to_morton_64bit(xrow + k, xcol + j);
                X[cur] = min(X[cur], X[first] + X[second]);
            }
        }
    }
}

void serial_B_par(unsigned long *X, unsigned long *U, unsigned long *V,
                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    for (int64_t t = n-1; t >= 0; t--) {
        for (int64_t i = t; i <= n-1; i++) {
            int64_t j = i - t;
            for (int64_t k = i; k <= n - 1; k++) {
                cout << "serial B first loop t,i,j,k " << t << ", " << i << ", " << j << ", " << k << endl;
                uint64_t cur = encode2D_to_morton_64bit(xrow + i, xcol + j);
                uint64_t first = encode2D_to_morton_64bit(urow + i, ucol + k);
                uint64_t second = encode2D_to_morton_64bit(xrow + k, xcol + j);
                cout << "serial B first loop cur=" << cur << ", first=" << first << ", second=" << second << endl;
                cout << "serial B first loop X[cur]=" << X[cur] << ", U[first]=" << U[first] << ", V[second]=" << V[second] << endl;
                X[cur] = min(X[cur], U[first] + X[second]);
            }
            for (int64_t k = 0; k <= j; k++) {
                cout << "serial B second loop t,i,j,k " << t << ", " << i << ", " << j << ", " << k << endl;
                uint64_t cur = encode2D_to_morton_64bit(xrow + i, xcol + j);
                uint64_t first = encode2D_to_morton_64bit(xrow + i, xcol + k);
                uint64_t second = encode2D_to_morton_64bit(vrow + k, vcol + j);
                cout << "serial B second loop cur=" << cur << ", first=" << first << ", second=" << second << endl;
                cout << "serial B second loop X[cur]=" << X[cur] << ", U[first]=" << U[first] << ", V[second]=" << V[second] << endl;
                X[cur] = min(X[cur], X[first] + V[second]);
            }
        }
    }
}

void serial_C_par(unsigned long *X, unsigned long *U, unsigned long *V,
                  uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    for (int64_t i = 0; i < n; i++) {
        for (int64_t j = 0; j < n; j++) {
            for (int64_t k = 0; k < n; k++) {
                uint64_t cur = encode2D_to_morton_64bit(xrow + i, xcol + j);
                uint64_t first = encode2D_to_morton_64bit(urow + i, ucol + k);
                uint64_t second = encode2D_to_morton_64bit(vrow + k, vcol + j);
                X[cur] = min(X[cur], U[first] + V[second]); 
            }
        }
    }
}

/*
 * Host RAM code
 */
void host_RAM_A_par(unsigned long *X,
                    uint64_t xrow, uint64_t xcol, uint64_t n)
{
    cout << "RAM A\n";
    serial_A_par(X, xrow, xcol, n);
}

void host_RAM_B_par(unsigned long *X, unsigned long *U, unsigned long *V,
                    uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    cout << "Inside RAM B\n";
    cout << xrow << xcol << urow << ucol << vrow << vcol << endl;
    cout << "RAM B X input\n";
    for(int i = 0; i < n*n; i++) {
        cout << X[i] << " ";
    }
    cout << endl;
    cout << "RAM B U input\n";
    for(int i = 0; i < n*n; i++) {
        cout << U[i] << " ";
    }
    cout << endl;
    cout << "RAM B V input\n";
    for(int i = 0; i < n*n; i++) {
        cout << V[i] << " ";
    }
    cout << endl; 
    serial_B_par(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, n);
    cout << "RAM B X output\n";
    for(int i = 0; i < n*n; i++) {
        cout << X[i] << " ";
    }
    cout << endl;
}

void host_RAM_C_par(unsigned long *X, unsigned long *U, unsigned long *V,
                    uint64_t xrow, uint64_t xcol, uint64_t urow, uint64_t ucol, uint64_t vrow, uint64_t vcol, uint64_t n)
{
    cout << "RAM C\n";
    serial_C_par(X, U, V, xrow, xcol, urow, ucol, vrow, vcol, n);
}

